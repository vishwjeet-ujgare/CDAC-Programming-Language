#include "hip/hip_runtime.h"
#include "calculation.h"

/*
 * CUDA Kernel Device code
 *
 * Perform calculation on d_d using constant d_v and place results into d_i (examples are multiplying by d_v, or dividing by d_v, etc.)
 */
__global__ void calculation(int *d_d, int *d_i, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        d_i[i] = d_d[i] * d_v;
    }
}

__host__ int *allocateRandomHostMemory(int numElements)
{
    size_t size = numElements * sizeof(int);

    // Allocate the host input vectors h_d
    int *h_d = (int *)malloc(size);

    // Verify that allocations succeeded
    if (h_d == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_d[i] = rand() % 255;
    }

    return h_d;
}

// Based heavily on https://www.gormanalysis.com/blog/reading-and-writing-csv-files-with-cpp/
// Presumes that there is no header in the csv file
__host__ std::tuple<int * , int>readCsv(std::string filename)
{
    std::vector<int> tempResult;
    // Create an input filestream
    std::ifstream myFile(filename);

    // Make sure the file is open
    if(!myFile.is_open()) throw std::runtime_error("Could not open file");

    // Helper vars
    std::string line, colname;
    int val;

    // Read data, line by line
    while(std::getline(myFile, line))
    {
        // Create a stringstream of the current line
        std::stringstream ss(line);
        
        // Extract each integer
        while(ss >> val){
            tempResult.push_back(val);
            // If the next token is a comma, ignore it and move on
            if(ss.peek() == ',') ss.ignore();
        }
    }

    // Close file
    myFile.close();
    int numElements = tempResult.size();
    int result[numElements];
    // Copy all elements of vector to array
    std::copy(tempResult.begin(), tempResult.end(), result);

    return {result, numElements};
}

__host__ std::tuple<int *, int *> allocateDeviceMemory(int numElements)
{
    // Allocate the device input vector A
    int *d_d = NULL;
    size_t size = numElements * sizeof(int);
    hipError_t err = hipMalloc(&d_d, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_d (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int *d_i;
    err = hipMalloc(&d_i, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_i (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return {d_d, d_i};
}

__host__ void copyFromHostToDevice(int h_v, int *h_d, int *d_d, int numElements)
{
    size_t size = numElements * sizeof(int);

    hipError_t err = hipMemcpy(d_d, h_d, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpyToSymbol(HIP_SYMBOL(d_v), &h_v, sizeof(int), 0, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy constant int d_v from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__host__ void executeKernel(int *d_d, int *d_i, int numElements, int threadsPerBlock)
{
    // Launch the search CUDA Kernel
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    calculation<<<blocksPerGrid, threadsPerBlock>>>(d_d, d_i, numElements);
    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__host__ void copyFromDeviceToHost(int *d_i, int *h_i, int numElements)
{
    cout << "Copying from Device to Host\n";
    // Copy the device result int array in device memory to the host result int array in host memory.
    size_t size = numElements * sizeof(int);

    hipError_t err = hipMemcpy(h_i, d_i, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy array d_i from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Free device global memory
__host__ void deallocateMemory(int *d_d, int *d_i)
{

    hipError_t err = hipFree(d_d);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_d (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_i);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device int variable d_i (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

}

// Reset the device and exit
__host__ void cleanUpDevice()
{
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipError_t err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/*
 * Host main routine
 */
int main(int argc, char *argv[])
{
    srand(time(0));
    int numElements = 10;
    int h_v = -1;
    int *h_d;
    int threadsPerBlock = 256;

    bool sortInputData = true;

    // calculation.exe true|false threadsPerBlock numElements searchValue inputFilename

    if(argc > 1)
    {
        std::string sortInputDataStr(argv[1]);
        if(sortInputDataStr == "false")
        {
            sortInputData = false;
        }
    }

    if(argc > 2)
    {
        threadsPerBlock = atoi(argv[2]);
        if(argc > 3)
        {
            numElements = atoi(argv[3]);
        }
    }
    if(argc > 4)
    {
        h_v = atoi(argv[4]);
        std::string inputFilename(argv[5]);
        tuple<int *, int>csvData = readCsv(inputFilename);
        h_d = get<0>(csvData);
        numElements = get<1>(csvData);
    }
    else 
    {
        h_d = allocateRandomHostMemory(numElements);
        // This is a bit hard coded, could consider coming up with another randomization scheme
        h_v = rand() % 255;
    }

    if(sortInputData)
    {
        sort(h_d, h_d + numElements);
    }

    int *h_i = (int *)malloc(numElements * sizeof(int));
    cout << "Data: ";
    for (int i = 0; i < numElements; ++i)
    {
        cout << h_d[i] << " ";
        h_i[i]=0;
    }
    cout << "\n";

    printf("Searching for value: %d \n", h_v);
    auto[d_d, d_i] = allocateDeviceMemory(numElements);
    copyFromHostToDevice(h_v, h_d, d_d, numElements);

    executeKernel(d_d, d_i, numElements, threadsPerBlock);

    copyFromDeviceToHost(d_i, h_i, numElements);

    cout << "Calculation results: ";
    for (int i = 0; i < numElements; ++i)
    {
        cout << h_i[i] << "\n";
    }

    deallocateMemory(d_d, d_i);

    cleanUpDevice();
    return 0;
}